#inlcude <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <nvtx3/nvToolsExt.h>

#include "./mykernels.cuh"

void invoke_cudnn_conv(float *d_inp, int M, int N, float *d_kernel, int m, int n, float *d_out, float &elapsed_time, float *h_output=NULL, bool copy_output = false, int measurement_iterations=1, int warmup_iterations=1) {
    cudaHandle_t handle;
    hipEvent_t beg, end;

    hipdnnCreate(&handle);
    hipEventCreate(&beg);
    hipEventCreate(&end);

    hipdnnTensorDescriptor_t inputDescriptor, outputDescriptor;
    hipdnnFilterDescriptor_t kernelDescriptor;
    hipdnnConvolutionDescriptor_t convolutionDescriptor;

    hipdnnCreateTensorDescriptor(&inputDescriptor);
    hipdnnCreateTensorDescriptor(&outputDescriptor);
    hipdnnCreateFilterDescriptor(&kernelDescriptor);
    hipdnnCreateConvolutionDescriptor(&convolutionDescriptor);

    int batchSize = 1, channels=1;
    int pad_m = m/2;
    int pad_n = n/2;
    int strid_m = 1, stride_n = 1;
    int dilation_m = 1, dilantion_n = 1;

    hipdnnSetTensor4dDescriptor(
        inputDescriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batchSize, channels, M, N
    );

    hipdnnSetFilter4dDescriptor(
        kernelDescriptor,
        HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW,
        1,1,m,n
    );

    hipdnnSetConvolution2dDescriptor(
        convolutionDescriptor,
        pad_m,pad_n,
        stride_m,stride_n,
        dilation_m,dilation_n,
        HIPDNN_CROSS_CORRELATION,
        HIPDNN_DATA_FLOAT
    );

    int out_M, out_N;
    hipdnnGetConvolution2dForwardOutputDim(
        convolutionDescriptor,
        inputDescriptor,
        kernelDescriptor,
        &batchSize, &channels,
        &out_M, &out_N
    );

    hipdnnSetTensor4dDescriptor(
        outputDescriptor,
        HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT,
        batchSize, channels, out_M, out_N
    );

    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(
        handle,
        inputDescriptor,
        kernelDescriptor,
        convolutionDescriptor,
        outputDescriptor,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
        0, &algo
    );

    size_t workspaceBytes = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle,
        inputDescriptor,
        kernelDescriptor,
        convolutionDescriptor,
        outputDescriptor,
        algo,
        &workspaceBytes
    );

    float *d_workspace = NULL;
    if(workspaceBytes > 0) MCC(hipMalloc(&d_workspace, workspaceBytes));

    float alpha=1.0f, beta=0.0f;

    hipdnnConvolutionForward(handle, &alpha, inputDescriptor, d_inp, 
        kernelDescriptor, d_kernel, convolutionDescriptor, algo,
        d_workspace, workspaceBytes, &beta, outputDescriptor, d_out
    );

    if(copy_output && h_output != NULL) MMC(hipMemcpy(h_output, d_out, sizeof(float)*out_M*out_N, hipMemcpyDeviceToHost));

    for(int i=0 ;i<warmup_iterations-1;i++){
        hipdnnConvolutionForward(handle, &alpha, inputDescriptor, d_inp, 
            kernelDescriptor, d_kernel, convolutionDescriptor, algo,
            d_workspace, workspaceBytes, &beta, outputDescriptor, d_out
        );
        hipDeviceSynchronize();
    }

    hipEventRecord(beg);
    nvtxRangePush("CUDNN kernel");
    for(int i=0; i<measurement_iterations; i++){
        hipdnnConvolutionForward(handle, &alpha, inputDescriptor, d_inp, 
            kernelDescriptor, d_kernel, convolutionDescriptor, algo,
            d_workspace, workspaceBytes, &beta, outputDescriptor, d_out
        );
        hipDeviceSynchronize();
    }
    nvtxRangePop();
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);

    if (workspaceBytes > 0) MMC(hipFree(d_workspace));
    MMC(hipdnnDestroyTensorDescriptor(inputDescriptor));
    MMC(hipdnnDestroyTensorDescriptor(outputDescriptor));
    MMC(hipdnnDestroyFilterDescriptor(kernelDescriptor));
    MMC(hipdnnDestroyConvolutionDescriptor(convolutionDescriptor));
    hipdnnDestroy(handle);
}