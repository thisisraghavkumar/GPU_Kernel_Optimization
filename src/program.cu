#include "./kernels/mykernels.cuh"


void printMatrix(const char *heading, const float* matrix, int m, int n) {
    const int width = 10;      // Column width
    const int precision = 4;   // Decimal precision

    std::cout<<heading<<":\n";
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            std::cout << std::setw(width) << std::fixed << std::setprecision(precision)
                      << matrix[i * n + j] << " ";
        }
        std::cout << std::endl;
    }
}

/*
 * Function to populate an array of floats with random values
 */
void populate_array(float *arr, int size, std::mt19937 &gen, std::uniform_real_distribution<float> &dis)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] = dis(gen);
    }
}

void CudaDeviceInfo()
{
    int deviceId;

    hipGetDevice(&deviceId);

    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, deviceId);
    std::cout << "Device ID                         : " << deviceId << std::endl;
    std::cout << "Name                              : " << props.name << std::endl;
    std::cout << "Compute Capability                : " << props.major << "." << props.minor << std::endl;
    std::cout << "Memory Bus Width                  : " << props.memoryBusWidth << std::endl;
    std::cout << "Max threads per block             : " << props.maxThreadsPerBlock << std::endl;
    std::cout << "Max threads per multi-processor   : " << props.maxThreadsPerMultiProcessor << std::endl;
    std::cout << "Registers per block               : " << props.regsPerBlock << std::endl;
    std::cout << "Registers per multi-processor     : " << props.regsPerMultiprocessor << std::endl;
    std::cout << "Total Global Memory               : " << props.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    std::cout << "Shared Memory per block           : " << props.sharedMemPerBlock / 1024 << "KB" << std::endl;
    std::cout << "Shared Memory per multi-processor : " << props.sharedMemPerMultiprocessor / 1024 << "KB" << std::endl;
    std::cout << "Total Constant Memory             : " << props.totalConstMem / 1024 << "KB" << std::endl;
    std::cout << "Multi-processor count             : " << props.multiProcessorCount << std::endl;
    std::cout << "Warp Size                         : " << props.warpSize << std::endl;
    std::cout << "----------------------------------------------------------------" << std::endl;
}

int main(){
    CudaDeviceInfo();

    int M = INPROW; // input height
    int N = INPCOL; // input width
    const int m = KERROW;    // kernel height
    const int n = KERCOL;    // kernel width

    int input_size = M*N;
    int kernel_size = m*n;

    int warmpup_runs = 5;
    int measurement_runs = 50;
    long long numoperations = M*N*(2*m*n - 1);
    float *h_inp, *h_fil, *h_out, *h_out_ref;
    float *d_inp, *d_fil, *d_out;

    float elapsed_time;
    hipEvent_t beg, end;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(50.0, 25.0);

    MCC(hipMalloc(&d_inp, sizeof(float)*input_size));
    MCC(hipMalloc(&d_fil, sizeof(float)*kernel_size)); // maybe I should use constant space in GPU for small kernels
    MCC(hipMalloc(&d_out, sizeof(float)*input_size));

    h_inp = new float[input_size];
    h_fil = new float[kernel_size];
    h_out = new float[input_size];
    h_out_ref = new float[input_size];
    populate_array(h_inp, input_size, gen, dis);
    //printMatrix("Input",h_inp, M, N);
    populate_array(h_fil, kernel_size, gen, dis);
    //printMatrix("Kernel", h_fil, m, n);
    MCC(hipMemcpy(d_inp, h_inp, sizeof(float)*input_size, hipMemcpyHostToDevice));
    MCC(hipMemcpy(d_fil, h_fil, sizeof(float)*kernel_size, hipMemcpyHostToDevice)); // maybe this will change if I choose to use constant space for small kernels
    //MCC(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel_const), h_fil, sizeof(float)*kernel_size));

    invoke_mynaivekernel(d_inp, M, N, d_fil, m, n, d_out, false);
    MCC(hipMemcpy(h_out_ref, d_out, sizeof(float)*input_size, hipMemcpyDeviceToHost));
    //printMatrix("Output with kernel in memory", h_out_ref, M, N);
    //invoke_mynaivekernel(d_inp, M, N, d_fil, m, n, d_out, true);
    //MCC(hipMemcpy(h_out_ref, d_out, sizeof(float)*input_size, hipMemcpyDeviceToHost));
    //printMatrix("Output with kernel as constant",h_out_ref, M, N);

    auto printRow = [](const std::string &name, float time, long long ops, int runs)
    {
        float avg_time = time / runs;
        float gflops = (ops / (avg_time / 1000.0f)) / 1e9;
        std::cout << std::setw(20) << std::left << name
                  << std::setw(20) << avg_time
                  << std::setw(20) << gflops << std::endl;
    };

    std::cout << std::fixed << std::setprecision(5);
    std::cout << "Number of operations: " << numoperations << std::endl;
    std::cout << std::fixed << std::setprecision(5); // Set decimal precision for floats

    // Print the table header
    std::cout << std::setw(20) << std::left << "Kernel Name"
              << std::setw(20) << "Time Taken (ms)"
              << std::setw(20) << "GFLOP/S" << std::endl;

    std::cout << std::string(60, '-') << std::endl;
    run_kernel("Naive kernel in memory", invoke_mynaivekernel, d_inp,M,N, d_fil, m, n, d_out, h_out, h_out_ref, &elapsed_time, gen, false, 5, measurement_runs);
    printRow("Naive kernel", elapsed_time, numoperations, measurement_runs);
    run_kernel("Shared memory kernel 1 with kernel in memory", invoke_mysharedmemkernel1, d_inp,M,N, d_fil, m, n, d_out, h_out, h_out_ref, &elapsed_time, gen, false, 5, measurement_runs);
    printRow("Shared memory kernel 1 with kernel in memory", elapsed_time, numoperations, measurement_runs);

    MCC(hipFree(d_inp));
    MCC(hipFree(d_fil));
    MCC(hipFree(d_out));
    free(h_inp);
    free(h_fil);
    free(h_out);
    free(h_out_ref);
}
